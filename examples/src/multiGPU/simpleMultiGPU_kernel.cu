/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * This application demonstrates how to use the CUDA API to use multiple GPUs.
 *
 * Note that in order to detect multiple GPUs in your system you have to disable
 * SLI in the nvidia control panel. Otherwise only one GPU is visible to the 
 * application. On the other side, you can still extend your desktop to screens 
 * attached to both GPUs.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

////////////////////////////////////////////////////////////////////////////////
// Simple reduction kernel.
// Refer to the 'reduction' CUDA SDK sample describing
// reduction optimization strategies
////////////////////////////////////////////////////////////////////////////////
__global__ static void reduceKernel(float *d_Result, float *d_Input, int N){
    const int     tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int threadN = gridDim.x * blockDim.x;
    float sum = 0;
    for(int pos = tid; pos < N; pos += threadN)
        sum += d_Input[pos];

    d_Result[tid] = sum;
}

extern "C" 
void launch_reduceKernel(float *d_Result, float *d_Input, int N, int BLOCK_N, int THREAD_N, hipStream_t s) 
{
	reduceKernel<<<BLOCK_N, THREAD_N, 0, s>>>(d_Result, d_Input, N);
}

